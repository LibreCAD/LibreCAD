// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cxx11_tensor_random_cuda
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU

#include "main.h"
#include <Eigen/CXX11/Tensor>


void test_cuda_random_uniform()
{
  Tensor<float, 2> out(72,97);
  out.setZero();

  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_out;
  hipMalloc((void**)(&d_out), out_bytes);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  gpu_out.device(gpu_device) = gpu_out.random();

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  // For now we just check thes code doesn't crash.
  // TODO: come up with a valid test of randomness
}


void test_cuda_random_normal()
{
  Tensor<float, 2> out(72,97);
  out.setZero();

  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_out;
  hipMalloc((void**)(&d_out), out_bytes);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  Eigen::internal::NormalRandomGenerator<float> gen(true);
  gpu_out.device(gpu_device) = gpu_out.random(gen);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);
}

static void test_complex()
{
  Tensor<std::complex<float>, 1> vec(6);
  vec.setRandom();

  // Fixme: we should check that the generated numbers follow a uniform
  // distribution instead.
  for (int i = 1; i < 6; ++i) {
    VERIFY_IS_NOT_EQUAL(vec(i), vec(i-1));
  }
}


void test_cxx11_tensor_random_cuda()
{
  CALL_SUBTEST(test_cuda_random_uniform());
  CALL_SUBTEST(test_cuda_random_normal());
  CALL_SUBTEST(test_complex());
}
