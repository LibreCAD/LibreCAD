#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cxx11_tensor_cuda
#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

void test_cuda_nullary() {
  Tensor<float, 1, 0, int> in1(2);
  Tensor<float, 1, 0, int> in2(2);
  in1.setRandom();
  in2.setRandom();

  std::size_t tensor_bytes = in1.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  hipMalloc((void**)(&d_in1), tensor_bytes);
  hipMalloc((void**)(&d_in2), tensor_bytes);
  hipMemcpy(d_in1, in1.data(), tensor_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), tensor_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in1(
      d_in1, 2);
  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in2(
      d_in2, 2);

  gpu_in1.device(gpu_device) = gpu_in1.constant(3.14f);
  gpu_in2.device(gpu_device) = gpu_in2.random();

  Tensor<float, 1, 0, int> new1(2);
  Tensor<float, 1, 0, int> new2(2);

  assert(hipMemcpyAsync(new1.data(), d_in1, tensor_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipMemcpyAsync(new2.data(), d_in2, tensor_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);

  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(new1(i), 3.14f);
    VERIFY_IS_NOT_EQUAL(new2(i), in2(i));
  }

  hipFree(d_in1);
  hipFree(d_in2);
}

void test_cuda_elementwise_small() {
  Tensor<float, 1> in1(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> in2(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> out(Eigen::array<Eigen::DenseIndex, 1>(2));
  in1.setRandom();
  in2.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in2(
      d_in2, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_out(
      d_out, Eigen::array<Eigen::DenseIndex, 1>(2));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(
        out(Eigen::array<Eigen::DenseIndex, 1>(i)),
        in1(Eigen::array<Eigen::DenseIndex, 1>(i)) + in2(Eigen::array<Eigen::DenseIndex, 1>(i)));
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
}

void test_cuda_elementwise()
{
  Tensor<float, 3> in1(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> in2(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> in3(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> out(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  in1.setRandom();
  in2.setRandom();
  in3.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t in3_bytes = in3.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_in3;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_in3), in3_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in3, in3.data(), in3_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in3(d_in3, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2 * gpu_in3;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 53; ++j) {
      for (int k = 0; k < 97; ++k) {
        VERIFY_IS_APPROX(out(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)), in1(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) + in2(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) * in3(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)));
      }
    }
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_in3);
  hipFree(d_out);
}

void test_cuda_props() {
  Tensor<float, 1> in1(200);
  Tensor<bool, 1> out(200);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(bool);

  float* d_in1;
  bool* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, 200);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_out(
      d_out, 200);

  gpu_out.device(gpu_device) = (gpu_in1.isnan)();

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 200; ++i) {
    VERIFY_IS_EQUAL(out(i), (std::isnan)(in1(i)));
  }

  hipFree(d_in1);
  hipFree(d_out);
}

void test_cuda_reduction()
{
  Tensor<float, 4> in1(72,53,97,113);
  Tensor<float, 2> out(72,97);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4> > gpu_in1(d_in1, 72,53,97,113);
  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  array<Eigen::DenseIndex, 2> reduction_axis;
  reduction_axis[0] = 1;
  reduction_axis[1] = 3;

  gpu_out.device(gpu_device) = gpu_in1.maximum(reduction_axis);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      float expected = 0;
      for (int k = 0; k < 53; ++k) {
        for (int l = 0; l < 113; ++l) {
          expected =
              std::max<float>(expected, in1(i, k, j, l));
        }
      }
      VERIFY_IS_APPROX(out(i,j), expected);
    }
  }

  hipFree(d_in1);
  hipFree(d_out);
}

template<int DataLayout>
void test_cuda_contraction()
{
  // with these dimensions, the output has 300 * 140 elements, which is
  // more than 30 * 1024, which is the number of threads in blocks on
  // a 15 SM GK110 GPU
  Tensor<float, 4, DataLayout> t_left(6, 50, 3, 31);
  Tensor<float, 5, DataLayout> t_right(Eigen::array<Eigen::DenseIndex, 5>(3, 31, 7, 20, 1));
  Tensor<float, 5, DataLayout> t_result(Eigen::array<Eigen::DenseIndex, 5>(6, 50, 7, 20, 1));

  t_left.setRandom();
  t_right.setRandom();

  std::size_t t_left_bytes = t_left.size()  * sizeof(float);
  std::size_t t_right_bytes = t_right.size() * sizeof(float);
  std::size_t t_result_bytes = t_result.size() * sizeof(float);

  float* d_t_left;
  float* d_t_right;
  float* d_t_result;

  hipMalloc((void**)(&d_t_left), t_left_bytes);
  hipMalloc((void**)(&d_t_right), t_right_bytes);
  hipMalloc((void**)(&d_t_result), t_result_bytes);

  hipMemcpy(d_t_left, t_left.data(), t_left_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_t_right, t_right.data(), t_right_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_t_left(d_t_left, 6, 50, 3, 31);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_right(d_t_right, 3, 31, 7, 20, 1);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_result(d_t_result, 6, 50, 7, 20, 1);

  typedef Eigen::Map<Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> > MapXf;
  MapXf m_left(t_left.data(), 300, 93);
  MapXf m_right(t_right.data(), 93, 140);
  Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> m_result(300, 140);

  typedef Tensor<float, 1>::DimensionPair DimPair;
  Eigen::array<DimPair, 2> dims;
  dims[0] = DimPair(2, 0);
  dims[1] = DimPair(3, 1);

  m_result = m_left * m_right;
  gpu_t_result.device(gpu_device) = gpu_t_left.contract(gpu_t_right, dims);

  hipMemcpy(t_result.data(), d_t_result, t_result_bytes, hipMemcpyDeviceToHost);

  for (DenseIndex i = 0; i < t_result.size(); i++) {
    if (fabs(t_result.data()[i] - m_result.data()[i]) >= 1e-4f) {
      std::cout << "mismatch detected at index " << i << ": " << t_result.data()[i] << " vs " <<  m_result.data()[i] << std::endl;
      assert(false);
    }
  }

  hipFree(d_t_left);
  hipFree(d_t_right);
  hipFree(d_t_result);
}

template<int DataLayout>
void test_cuda_convolution_1d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 1, DataLayout> kernel(4);
  Tensor<float, 4, DataLayout> out(74,34,11,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input, 74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 1, DataLayout> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out, 74,34,11,137);

  Eigen::array<Eigen::DenseIndex, 1> dims(1);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 34; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k,l) * kernel(0) + input(i,j+1,k,l) * kernel(1) +
                                 input(i,j+2,k,l) * kernel(2) + input(i,j+3,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

void test_cuda_convolution_inner_dim_col_major_1d()
{
  Tensor<float, 4, ColMajor> input(74,9,11,7);
  Tensor<float, 1, ColMajor> kernel(4);
  Tensor<float, 4, ColMajor> out(71,9,11,7);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_input(d_input,74,9,11,7);
  Eigen::TensorMap<Eigen::Tensor<float, 1, ColMajor> > gpu_kernel(d_kernel,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_out(d_out,71,9,11,7);

  Eigen::array<Eigen::DenseIndex, 1> dims(0);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 71; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 7; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i+0,j,k,l) * kernel(0) + input(i+1,j,k,l) * kernel(1) +
                                 input(i+2,j,k,l) * kernel(2) + input(i+3,j,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

void test_cuda_convolution_inner_dim_row_major_1d()
{
  Tensor<float, 4, RowMajor> input(7,9,11,74);
  Tensor<float, 1, RowMajor> kernel(4);
  Tensor<float, 4, RowMajor> out(7,9,11,71);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_input(d_input, 7,9,11,74);
  Eigen::TensorMap<Eigen::Tensor<float, 1, RowMajor> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_out(d_out, 7,9,11,71);

  Eigen::array<Eigen::DenseIndex, 1> dims(3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 7; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 71; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j,k,l+0) * kernel(0) + input(i,j,k,l+1) * kernel(1) +
                                 input(i,j,k,l+2) * kernel(2) + input(i,j,k,l+3) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

template<int DataLayout>
void test_cuda_convolution_2d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 2, DataLayout> kernel(3,4);
  Tensor<float, 4, DataLayout> out(74,35,8,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input,74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> > gpu_kernel(d_kernel,3,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out,74,35,8,137);

  Eigen::array<Eigen::DenseIndex, 2> dims(1,2);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k+0,l) * kernel(0,0) +
                                 input(i,j+1,k+0,l) * kernel(1,0) +
                                 input(i,j+2,k+0,l) * kernel(2,0) +
                                 input(i,j+0,k+1,l) * kernel(0,1) +
                                 input(i,j+1,k+1,l) * kernel(1,1) +
                                 input(i,j+2,k+1,l) * kernel(2,1) +
                                 input(i,j+0,k+2,l) * kernel(0,2) +
                                 input(i,j+1,k+2,l) * kernel(1,2) +
                                 input(i,j+2,k+2,l) * kernel(2,2) +
                                 input(i,j+0,k+3,l) * kernel(0,3) +
                                 input(i,j+1,k+3,l) * kernel(1,3) +
                                 input(i,j+2,k+3,l) * kernel(2,3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}

template<int DataLayout>
void test_cuda_convolution_3d()
{
  Tensor<float, 5, DataLayout> input(Eigen::array<Eigen::DenseIndex, 5>(74,37,11,137,17));
  Tensor<float, 3, DataLayout> kernel(3,4,2);
  Tensor<float, 5, DataLayout> out(Eigen::array<Eigen::DenseIndex, 5>(74,35,8,136,17));
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  hipMalloc((void**)(&d_input), input_bytes);
  hipMalloc((void**)(&d_kernel), kernel_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_input, input.data(), input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel.data(), kernel_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;    
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_input(d_input,74,37,11,137,17);
  Eigen::TensorMap<Eigen::Tensor<float, 3, DataLayout> > gpu_kernel(d_kernel,3,4,2);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_out(d_out,74,35,8,136,17);

  Eigen::array<Eigen::DenseIndex, 3> dims(1,2,3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 136; ++l) {
          for (int m = 0; m < 17; ++m) {
            const float result = out(i,j,k,l,m);
            const float expected = input(i,j+0,k+0,l+0,m) * kernel(0,0,0) +
                                   input(i,j+1,k+0,l+0,m) * kernel(1,0,0) +
                                   input(i,j+2,k+0,l+0,m) * kernel(2,0,0) +
                                   input(i,j+0,k+1,l+0,m) * kernel(0,1,0) +
                                   input(i,j+1,k+1,l+0,m) * kernel(1,1,0) +
                                   input(i,j+2,k+1,l+0,m) * kernel(2,1,0) +
                                   input(i,j+0,k+2,l+0,m) * kernel(0,2,0) +
                                   input(i,j+1,k+2,l+0,m) * kernel(1,2,0) +
                                   input(i,j+2,k+2,l+0,m) * kernel(2,2,0) +
                                   input(i,j+0,k+3,l+0,m) * kernel(0,3,0) +
                                   input(i,j+1,k+3,l+0,m) * kernel(1,3,0) +
                                   input(i,j+2,k+3,l+0,m) * kernel(2,3,0) +
                                   input(i,j+0,k+0,l+1,m) * kernel(0,0,1) +
                                   input(i,j+1,k+0,l+1,m) * kernel(1,0,1) +
                                   input(i,j+2,k+0,l+1,m) * kernel(2,0,1) +
                                   input(i,j+0,k+1,l+1,m) * kernel(0,1,1) +
                                   input(i,j+1,k+1,l+1,m) * kernel(1,1,1) +
                                   input(i,j+2,k+1,l+1,m) * kernel(2,1,1) +
                                   input(i,j+0,k+2,l+1,m) * kernel(0,2,1) +
                                   input(i,j+1,k+2,l+1,m) * kernel(1,2,1) +
                                   input(i,j+2,k+2,l+1,m) * kernel(2,2,1) +
                                   input(i,j+0,k+3,l+1,m) * kernel(0,3,1) +
                                   input(i,j+1,k+3,l+1,m) * kernel(1,3,1) +
                                   input(i,j+2,k+3,l+1,m) * kernel(2,3,1);
            VERIFY_IS_APPROX(result, expected);
          }
        }
      }
    }
  }

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_out);
}


template <typename Scalar>
void test_cuda_lgamma(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  hipMalloc((void**)(&d_in), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.lgamma();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::lgamma)(in(i,j)));
    }
  }

  hipFree(d_in);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_digamma()
{
  Tensor<Scalar, 1> in(7);
  Tensor<Scalar, 1> out(7);
  Tensor<Scalar, 1> expected_out(7);
  out.setZero();

  in(0) = Scalar(1);
  in(1) = Scalar(1.5);
  in(2) = Scalar(4);
  in(3) = Scalar(-10.5);
  in(4) = Scalar(10000.5);
  in(5) = Scalar(0);
  in(6) = Scalar(-1);

  expected_out(0) = Scalar(-0.5772156649015329);
  expected_out(1) = Scalar(0.03648997397857645);
  expected_out(2) = Scalar(1.2561176684318);
  expected_out(3) = Scalar(2.398239129535781);
  expected_out(4) = Scalar(9.210340372392849);
  expected_out(5) = std::numeric_limits<Scalar>::infinity();
  expected_out(6) = std::numeric_limits<Scalar>::infinity();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  hipMalloc((void**)(&d_in), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 7);

  gpu_out.device(gpu_device) = gpu_in.digamma();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 5; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }
  for (int i = 5; i < 7; ++i) {
    VERIFY_IS_EQUAL(out(i), expected_out(i));
  }

  hipFree(d_in);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_zeta()
{
  Tensor<Scalar, 1> in_x(6);
  Tensor<Scalar, 1> in_q(6);
  Tensor<Scalar, 1> out(6);
  Tensor<Scalar, 1> expected_out(6);
  out.setZero();

  in_x(0) = Scalar(1);
  in_x(1) = Scalar(1.5);
  in_x(2) = Scalar(4);
  in_x(3) = Scalar(-10.5);
  in_x(4) = Scalar(10000.5);
  in_x(5) = Scalar(3);
  
  in_q(0) = Scalar(1.2345);
  in_q(1) = Scalar(2);
  in_q(2) = Scalar(1.5);
  in_q(3) = Scalar(3);
  in_q(4) = Scalar(1.0001);
  in_q(5) = Scalar(-2.5);

  expected_out(0) = std::numeric_limits<Scalar>::infinity();
  expected_out(1) = Scalar(1.61237534869);
  expected_out(2) = Scalar(0.234848505667);
  expected_out(3) = Scalar(1.03086757337e-5);
  expected_out(4) = Scalar(0.367879440865);
  expected_out(5) = Scalar(0.054102025820864097);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_q;
  Scalar* d_out;
  hipMalloc((void**)(&d_in_x), bytes);
  hipMalloc((void**)(&d_in_q), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in_x, in_x.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in_q, in_q.data(), bytes, hipMemcpyHostToDevice);
  
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_q(d_in_q, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 6);

  gpu_out.device(gpu_device) = gpu_in_x.zeta(gpu_in_q);

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  VERIFY_IS_EQUAL(out(0), expected_out(0));
  VERIFY((std::isnan)(out(3)));

  for (int i = 1; i < 6; ++i) {
    if (i != 3) {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  hipFree(d_in_x);
  hipFree(d_in_q);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_polygamma()
{
  Tensor<Scalar, 1> in_x(7);
  Tensor<Scalar, 1> in_n(7);
  Tensor<Scalar, 1> out(7);
  Tensor<Scalar, 1> expected_out(7);
  out.setZero();

  in_n(0) = Scalar(1);
  in_n(1) = Scalar(1);
  in_n(2) = Scalar(1);
  in_n(3) = Scalar(17);
  in_n(4) = Scalar(31);
  in_n(5) = Scalar(28);
  in_n(6) = Scalar(8);
  
  in_x(0) = Scalar(2);
  in_x(1) = Scalar(3);
  in_x(2) = Scalar(25.5);
  in_x(3) = Scalar(4.7);
  in_x(4) = Scalar(11.8);
  in_x(5) = Scalar(17.7);
  in_x(6) = Scalar(30.2);

  expected_out(0) = Scalar(0.644934066848);
  expected_out(1) = Scalar(0.394934066848);
  expected_out(2) = Scalar(0.0399946696496);
  expected_out(3) = Scalar(293.334565435);
  expected_out(4) = Scalar(0.445487887616);
  expected_out(5) = Scalar(-2.47810300902e-07);
  expected_out(6) = Scalar(-8.29668781082e-09);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_n;
  Scalar* d_out;
  hipMalloc((void**)(&d_in_x), bytes);
  hipMalloc((void**)(&d_in_n), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in_x, in_x.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in_n, in_n.data(), bytes, hipMemcpyHostToDevice);
  
  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_n(d_in_n, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 7);

  gpu_out.device(gpu_device) = gpu_in_n.polygamma(gpu_in_x);

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 7; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  hipFree(d_in_x);
  hipFree(d_in_n);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_igamma()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igamma_s[][6] = {{0.0, nan, nan, nan, nan, nan},
                          {0.0, 0.6321205588285578, 0.7768698398515702,
                           0.9816843611112658, 9.999500016666262e-05, 1.0},
                          {0.0, 0.4275932955291202, 0.608374823728911,
                           0.9539882943107686, 7.522076445089201e-07, 1.0},
                          {0.0, 0.01898815687615381, 0.06564245437845008,
                           0.5665298796332909, 4.166333347221828e-18, 1.0},
                          {0.0, 0.9999780593618628, 0.9999899967080838,
                           0.9999996219837988, 0.9991370418689945, 1.0},
                          {0.0, 0.0, 0.0, 0.0, 0.0, 0.5042041932513908}};



  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  assert(hipMalloc((void**)(&d_a), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_x), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_out), bytes) == hipSuccess);

  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_x, x.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igamma(gpu_x);

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igamma_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igamma_s[i][j]);
      }
    }
  }

  hipFree(d_a);
  hipFree(d_x);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_igammac()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igammac_s[][6] = {{nan, nan, nan, nan, nan, nan},
                           {1.0, 0.36787944117144233, 0.22313016014842982,
                            0.018315638888734182, 0.9999000049998333, 0.0},
                           {1.0, 0.5724067044708798, 0.3916251762710878,
                            0.04601170568923136, 0.9999992477923555, 0.0},
                           {1.0, 0.9810118431238462, 0.9343575456215499,
                            0.4334701203667089, 1.0, 0.0},
                           {1.0, 2.1940638138146658e-05, 1.0003291916285e-05,
                            3.7801620118431334e-07, 0.0008629581310054535,
                            0.0},
                           {1.0, 1.0, 1.0, 1.0, 1.0, 0.49579580674813944}};

  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  hipMalloc((void**)(&d_a), bytes);
  hipMalloc((void**)(&d_x), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_x, x.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igammac(gpu_x);

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igammac_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igammac_s[i][j]);
      }
    }
  }

  hipFree(d_a);
  hipFree(d_x);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_erf(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  assert(hipMalloc((void**)(&d_in), bytes) == hipSuccess);
  assert(hipMalloc((void**)(&d_out), bytes) == hipSuccess);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erf();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erf)(in(i,j)));
    }
  }

  hipFree(d_in);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_erfc(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  hipMalloc((void**)(&d_in), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in, in.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erfc();

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erfc)(in(i,j)));
    }
  }

  hipFree(d_in);
  hipFree(d_out);
}

template <typename Scalar>
void test_cuda_betainc()
{
  Tensor<Scalar, 1> in_x(125);
  Tensor<Scalar, 1> in_a(125);
  Tensor<Scalar, 1> in_b(125);
  Tensor<Scalar, 1> out(125);
  Tensor<Scalar, 1> expected_out(125);
  out.setZero();

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();

  Array<Scalar, 1, Dynamic> x(125);
  Array<Scalar, 1, Dynamic> a(125);
  Array<Scalar, 1, Dynamic> b(125);
  Array<Scalar, 1, Dynamic> v(125);

  a << 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999,
      0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999,
      0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999;

  b << 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 999.999, 999.999, 999.999, 999.999, 999.999, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 999.999, 999.999, 999.999, 999.999, 999.999, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999;

  x << -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8,
      1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5,
      0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2,
      0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1,
      0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1,
      -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8,
      1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5,
      0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2,
      0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1;

  v << nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan,
      nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan,
      nan, nan, 0.47972119876364683, 0.5, 0.5202788012363533, nan, nan,
      0.9518683957740043, 0.9789663010413743, 0.9931729188073435, nan, nan,
      0.999995949033062, 0.9999999999993698, 0.9999999999999999, nan, nan,
      0.9999999999999999, 0.9999999999999999, 0.9999999999999999, nan, nan, nan,
      nan, nan, nan, nan, 0.006827081192655869, 0.0210336989586256,
      0.04813160422599567, nan, nan, 0.20014344256217678, 0.5000000000000001,
      0.7998565574378232, nan, nan, 0.9991401428435834, 0.999999999698403,
      0.9999999999999999, nan, nan, 0.9999999999999999, 0.9999999999999999,
      0.9999999999999999, nan, nan, nan, nan, nan, nan, nan,
      1.0646600232370887e-25, 6.301722877826246e-13, 4.050966937974938e-06, nan,
      nan, 7.864342668429763e-23, 3.015969667594166e-10, 0.0008598571564165444,
      nan, nan, 6.031987710123844e-08, 0.5000000000000007, 0.9999999396801229,
      nan, nan, 0.9999999999999999, 0.9999999999999999, 0.9999999999999999, nan,
      nan, nan, nan, nan, nan, nan, 0.0, 7.029920380986636e-306,
      2.2450728208591345e-101, nan, nan, 0.0, 9.275871147869727e-302,
      1.2232913026152827e-97, nan, nan, 0.0, 3.0891393081932924e-252,
      2.9303043666183996e-60, nan, nan, 2.248913486879199e-196,
      0.5000000000004947, 0.9999999999999999, nan;

  for (int i = 0; i < 125; ++i) {
    in_x(i) = x(i);
    in_a(i) = a(i);
    in_b(i) = b(i);
    expected_out(i) = v(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_a;
  Scalar* d_in_b;
  Scalar* d_out;
  hipMalloc((void**)(&d_in_x), bytes);
  hipMalloc((void**)(&d_in_a), bytes);
  hipMalloc((void**)(&d_in_b), bytes);
  hipMalloc((void**)(&d_out), bytes);

  hipMemcpy(d_in_x, in_x.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in_a, in_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in_b, in_b.data(), bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_a(d_in_a, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_b(d_in_b, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 125);

  gpu_out.device(gpu_device) = betainc(gpu_in_a, gpu_in_b, gpu_in_x);

  assert(hipMemcpyAsync(out.data(), d_out, bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 1; i < 125; ++i) {
    if ((std::isnan)(expected_out(i))) {
      VERIFY((std::isnan)(out(i)));
    } else {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  hipFree(d_in_x);
  hipFree(d_in_a);
  hipFree(d_in_b);
  hipFree(d_out);
}


void test_cxx11_tensor_cuda()
{
  CALL_SUBTEST_1(test_cuda_nullary());
  CALL_SUBTEST_1(test_cuda_elementwise_small());
  CALL_SUBTEST_1(test_cuda_elementwise());
  CALL_SUBTEST_1(test_cuda_props());
  CALL_SUBTEST_1(test_cuda_reduction());
  CALL_SUBTEST_2(test_cuda_contraction<ColMajor>());
  CALL_SUBTEST_2(test_cuda_contraction<RowMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_1d<ColMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_1d<RowMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_inner_dim_col_major_1d());
  CALL_SUBTEST_3(test_cuda_convolution_inner_dim_row_major_1d());
  CALL_SUBTEST_3(test_cuda_convolution_2d<ColMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_2d<RowMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_3d<ColMajor>());
  CALL_SUBTEST_3(test_cuda_convolution_3d<RowMajor>());

#if __cplusplus > 199711L
  // std::erf, std::erfc, and so on where only added in c++11. We use them
  // as a golden reference to validate the results produced by Eigen. Therefore
  // we can only run these tests if we use a c++11 compiler.
  CALL_SUBTEST_4(test_cuda_lgamma<float>(1.0f));
  CALL_SUBTEST_4(test_cuda_lgamma<float>(100.0f));
  CALL_SUBTEST_4(test_cuda_lgamma<float>(0.01f));
  CALL_SUBTEST_4(test_cuda_lgamma<float>(0.001f));

  CALL_SUBTEST_4(test_cuda_lgamma<double>(1.0));
  CALL_SUBTEST_4(test_cuda_lgamma<double>(100.0));
  CALL_SUBTEST_4(test_cuda_lgamma<double>(0.01));
  CALL_SUBTEST_4(test_cuda_lgamma<double>(0.001));

  CALL_SUBTEST_4(test_cuda_erf<float>(1.0f));
  CALL_SUBTEST_4(test_cuda_erf<float>(100.0f));
  CALL_SUBTEST_4(test_cuda_erf<float>(0.01f));
  CALL_SUBTEST_4(test_cuda_erf<float>(0.001f));

  CALL_SUBTEST_4(test_cuda_erfc<float>(1.0f));
  // CALL_SUBTEST(test_cuda_erfc<float>(100.0f));
  CALL_SUBTEST_4(test_cuda_erfc<float>(5.0f)); // CUDA erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_cuda_erfc<float>(0.01f));
  CALL_SUBTEST_4(test_cuda_erfc<float>(0.001f));

  CALL_SUBTEST_4(test_cuda_erf<double>(1.0));
  CALL_SUBTEST_4(test_cuda_erf<double>(100.0));
  CALL_SUBTEST_4(test_cuda_erf<double>(0.01));
  CALL_SUBTEST_4(test_cuda_erf<double>(0.001));

  CALL_SUBTEST_4(test_cuda_erfc<double>(1.0));
  // CALL_SUBTEST(test_cuda_erfc<double>(100.0));
  CALL_SUBTEST_4(test_cuda_erfc<double>(5.0)); // CUDA erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_cuda_erfc<double>(0.01));
  CALL_SUBTEST_4(test_cuda_erfc<double>(0.001));

  CALL_SUBTEST_5(test_cuda_digamma<float>());
  CALL_SUBTEST_5(test_cuda_digamma<double>());

  CALL_SUBTEST_5(test_cuda_polygamma<float>());
  CALL_SUBTEST_5(test_cuda_polygamma<double>());

  CALL_SUBTEST_5(test_cuda_zeta<float>());
  CALL_SUBTEST_5(test_cuda_zeta<double>());

  CALL_SUBTEST_5(test_cuda_igamma<float>());
  CALL_SUBTEST_5(test_cuda_igammac<float>());

  CALL_SUBTEST_5(test_cuda_igamma<double>());
  CALL_SUBTEST_5(test_cuda_igammac<double>());

  CALL_SUBTEST_6(test_cuda_betainc<float>());
  CALL_SUBTEST_6(test_cuda_betainc<double>());
#endif
}
